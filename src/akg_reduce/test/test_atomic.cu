#include "hip/hip_runtime.h"
#include "../utils/util.cuh"
#include "../operators/reduce_operators.cuh"
#include <cstdio>

using namespace akg_reduce;
using namespace std;

// check whether the op-atomic transformation is correct.
// compile code: nvcc test_atomic.cu -arch=sm_70

template <typename T>
__global__ void AtomicTestSum(T *dest, T val) {
  SumOp<T> op;
  AtomicOp<T, op.identifier> atomic_op;
  atomic_op.Compute(&dest[threadIdx.x], val);
}

template <typename T>
__global__ void AtomicTestMax(T *dest, T val) {
  MaxOp<T> op;
  AtomicOp<T, op.identifier> atomic_op;
  atomic_op.Compute(&dest[threadIdx.x], val);
}

template <typename T>
__global__ void AtomicTestMin(T *dest, T val) {
  MinOp<T> op;
  AtomicOp<T, op.identifier> atomic_op;
  atomic_op.Compute(&dest[threadIdx.x], val);
}

template <typename T>
void TestAtomicSum() {
  cout << "TestAtomicSum" << endl;
  int items = 1000;
  int bytes = items * sizeof(T);
  T *h_a, *d_a;

  h_a = (T *)malloc(bytes);
  for (auto i = 0; i < items; i++) {
    if (sizeof(T) == 2) {
      h_a[i] = __float2half(0.0);
    } else {
      h_a[i] = 0.0;
    }
  }

  GetGpuErr(hipMalloc((void **)&d_a, bytes));
  GetGpuErr(hipMemcpy((void *)d_a, (void *)h_a, bytes, hipMemcpyHostToDevice));

  dim3 grid(1000);
  dim3 block(1000);
  AtomicTestSum<T><<<grid, block>>>(d_a, 1.0);
  GetGpuErr(hipPeekAtLastError());

  GetGpuErr(hipMemcpy((void *)h_a, (void *)d_a, bytes, hipMemcpyDeviceToHost));

  for (auto i = 0; i < 10; i++) {
    double tmp;
    if (sizeof(T) == 2) {
      tmp = __half2float(h_a[i]);
    } else {
      tmp = h_a[i];
    }
    printf("%f ", tmp);
  }
  printf("\n");

  GetGpuErr(hipFree(d_a));
  free(h_a);
}

template <typename T>
void TestAtomicMax() {
  cout << "TestAtomicMax" << endl;
  int items = 10;
  int bytes = items * sizeof(T);
  T *h_a, *d_a;

  h_a = (T *)malloc(bytes);
  for (auto i = 0; i < items; i++) {
    if (sizeof(T) == 2) {
      h_a[i] = __float2half(i);
    } else {
      h_a[i] = i;
    }
  }

  GetGpuErr(hipMalloc((void **)&d_a, bytes));
  GetGpuErr(hipMemcpy((void *)d_a, (void *)h_a, bytes, hipMemcpyHostToDevice));

  double val = 1.234567891012345;
  dim3 grid(10000);
  dim3 block(items);
  AtomicTestMax<T><<<grid, block>>>(d_a, val);
  GetGpuErr(hipPeekAtLastError());

  GetGpuErr(hipMemcpy((void *)h_a, (void *)d_a, bytes, hipMemcpyDeviceToHost));

  for (auto i = 0; i < 5; i++) {
    double tmp;
    if (sizeof(T) == 2) {
      tmp = __half2float(h_a[i]);
    } else {
      tmp = h_a[i];
    }
    printf("%.12f ", tmp);
  }
  printf("\n");

  GetGpuErr(hipFree(d_a));
  free(h_a);
}

template <typename T>
void TestAtomicMin() {
  cout << "TestAtomicMin" << endl;
  int items = 10;
  int bytes = items * sizeof(T);
  T *h_a, *d_a;

  h_a = (T *)malloc(bytes);
  for (auto i = 0; i < items; i++) {
    h_a[i] = __float2half(i);
  }

  GetGpuErr(hipMalloc((void **)&d_a, bytes));
  GetGpuErr(hipMemcpy((void *)d_a, (void *)h_a, bytes, hipMemcpyHostToDevice));

  double val = 1.234567891012345;
  dim3 grid(10000);
  dim3 block(items);
  AtomicTestMin<T><<<grid, block>>>(d_a, val);
  GetGpuErr(hipPeekAtLastError());

  GetGpuErr(hipMemcpy((void *)h_a, (void *)d_a, bytes, hipMemcpyDeviceToHost));

  for (auto i = 0; i < 5; i++) {
    double tmp;
    if (sizeof(T) == 2) {
      tmp = __half2float(h_a[i]);
    } else {
      tmp = h_a[i];
    }
    printf("%.12f ", tmp);
  }
  printf("\n");

  GetGpuErr(hipFree(d_a));
  free(h_a);
}

int main() {
  TestAtomicSum<float>();
  TestAtomicSum<double>();
  TestAtomicSum<half>();

  TestAtomicMax<float>();
  TestAtomicMax<double>();
  TestAtomicMax<half>();

  TestAtomicMin<float>();
  TestAtomicMin<double>();
  TestAtomicMin<half>();

  return 0;
}
