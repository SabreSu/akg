#include "../utils/util.cuh"
using namespace akg_reduce;
using namespace std;

int main() {
  int items = 100;
  int bytes = items * sizeof(float);
  float *h_I, *d_I;
  h_I = (float *)malloc(bytes);

  GetGpuErr(hipMalloc((void **)&d_I, bytes));
  // check if GetGpuErr can detect and return properly.
  GetGpuErr(hipMemcpy((void *)d_I, (void *)h_I, bytes + 99, hipMemcpyHostToDevice));
  GetGpuErr(hipFree(d_I));
  free(h_I);

  return 0;
}
